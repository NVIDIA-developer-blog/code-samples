#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <stdexcept>
#include <vector>


#include <hipblas.h>
#include <hip/hip_runtime.h>

// CUDA API error checking
#define CUDA_CHECK(err)                                                                            \
    do {                                                                                           \
        hipError_t err_ = (err);                                                                  \
        if (err_ != hipSuccess) {                                                                 \
            std::printf("CUDA error %d at %s:%d\n", err_, __FILE__, __LINE__);                     \
            throw std::runtime_error("CUDA error");                                                \
        }                                                                                          \
    } while (0)

// cublas API error checking
#define CUBLAS_CHECK(err)                                                                          \
    do {                                                                                           \
        hipblasStatus_t err_ = (err);                                                               \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                                       \
            std::printf("cublas error %d at %s:%d\n", err_, __FILE__, __LINE__);                   \
            throw std::runtime_error("cublas error");                                              \
        }                                                                                          \
    } while (0)

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    hipEvent_t startEvent { nullptr };
    hipEvent_t stopEvent { nullptr };
    float       elapsed_gpu_ms {};
    
    int m, n, k;
    int lda, ldb, ldc;
    double alpha, beta;

    printf ("\n This example computes real matrix C=alpha*A*B+beta*C using \n"
            " cuBLAS dgemm, where A, B, and  C are matrices and \n"
            " alpha and beta are double precision scalars\n\n");

    int size = 4092;
    m = size, k = size, n = size;
    lda = size, ldb = size, ldc = size;
    printf (" Initializing data for matrix multiplication C=A*B for matrix \n"
        " A(%ix%i) and matrix B(%ix%i)\n\n", m, k, k, n);
    alpha = 1.0, beta = 0.0;

    printf (" Allocating memory for matrices aligned on 64-byte boundary for better \n"
            " performance \n\n");

    double *d_A = nullptr;
    double *d_B = nullptr;
    double *d_C = nullptr;

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * m * k));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(double) * k * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(double) * m * n));

    /* step 3: compute */
    printf (" Computing matrix product using cuBLAS dgemm function \n\n");

    hipEventCreateWithFlags( &startEvent, hipEventBlockingSync );
    hipEventRecord( startEvent );

    for (int i =0; i< 10; i++)
    CUBLAS_CHECK(
        hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, lda, d_B, ldb, &beta, d_C, ldc));

    CUDA_CHECK(hipStreamSynchronize(stream));

    hipEventCreateWithFlags( &stopEvent, hipEventBlockingSync );
    hipEventRecord( stopEvent );
    hipEventSynchronize( stopEvent );

    printf ("\n Computations completed.\n\n");

    hipEventElapsedTime( &elapsed_gpu_ms, startEvent, stopEvent );
    printf( " Time Elapsed: %0.2f ms \n\n", elapsed_gpu_ms/10);

    /* free resources */
    printf ("\n Deallocating memory \n\n");
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    printf (" Example completed. \n\n");
    return 0;
}
