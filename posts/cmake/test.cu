#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include "particle.h"
#include <stdlib.h>
#include <stdio.h>

__global__ void advanceParticles(float dt, particle * pArray, int nParticles)
{
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nParticles)
	{
		pArray[idx].advance(dt);
	}
}

int main(int argc, char ** argv)
{
	hipError_t error;
	int n = 1000000;
	if(argc > 1)	{ n = atoi(argv[1]);}     // Number of particles
	if(argc > 2)	{	srand(atoi(argv[2])); } // Random seed

	error = hipGetLastError();
	if (error != hipSuccess)
  	{
  	printf("0 %s\n",hipGetErrorString(error));
  	exit(1);
  	}

	particle * pArray = new particle[n];
	particle * devPArray = NULL;
	hipMalloc(&devPArray, n*sizeof(particle));
	hipDeviceSynchronize(); error = hipGetLastError();
	if (error != hipSuccess)
  	{
  	printf("1 %s\n",hipGetErrorString(error));
  	exit(1);
  	}

	hipMemcpy(devPArray, pArray, n*sizeof(particle), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); error = hipGetLastError();
	if (error != hipSuccess)
  	{
  	printf("2 %s\n",hipGetErrorString(error));
  	exit(1);
  	}

	for(int i=0; i<100; i++)
	{
		float dt = (float)rand()/(float) RAND_MAX; // Random distance each step
		advanceParticles<<< 1 +  n/256, 256>>>(dt, devPArray, n);
		error = hipGetLastError();
		if (error != hipSuccess)
    	{
    	printf("3 %s\n",hipGetErrorString(error));
    	exit(1);
    	}

		hipDeviceSynchronize();
	}
	hipMemcpy(pArray, devPArray, n*sizeof(particle), hipMemcpyDeviceToHost);

	v3 totalDistance(0,0,0);
	v3 temp;
	for(int i=0; i<n; i++)
	{
		temp = pArray[i].getTotalDistance();
		totalDistance.x += temp.x;
		totalDistance.y += temp.y;
		totalDistance.z += temp.z;
	}
	float avgX = totalDistance.x /(float)n;
	float avgY = totalDistance.y /(float)n;
	float avgZ = totalDistance.z /(float)n;
	float avgNorm = sqrt(avgX*avgX + avgY*avgY + avgZ*avgZ);
	printf(	"Moved %d particles 100 steps. Average distance traveled is |(%f, %f, %f)| = %f\n",
					n, avgX, avgY, avgZ, avgNorm);
	return 0;
}
